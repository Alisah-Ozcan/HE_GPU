﻿// --------------------- //
// Author: Alisah Ozcan
// --------------------- //

#include "context.cuh"


__host__ Parameters::Parameters(std::string scheme_type, int poly_degree, PrimePool::security_level sec_level) {

    if (scheme_type != "BFV")
        throw("Invalid Scheme Type");

    scheme = scheme_type;
    n = poly_degree;
    n_power = int(log2l(n));
    sec = sec_level;

    PrimePool pool(n, sec);
    coeff_modulus = pool.prime_count();
    bsk_modulus = pool.base_Bsk().size();
    total_bits = pool.total_primes_bits();


    hipMalloc(&modulus_, coeff_modulus * sizeof(Modulus));
    hipMemcpy(modulus_, pool.base_modulus().data(), coeff_modulus * sizeof(Modulus), hipMemcpyHostToDevice);

    hipMalloc(&ntt_table_, coeff_modulus * n * sizeof(Root));
    hipMemcpy(ntt_table_, pool.ntt_tables().data(), coeff_modulus * n * sizeof(Root), hipMemcpyHostToDevice);

    hipMalloc(&intt_table_, coeff_modulus * n * sizeof(Root));
    hipMemcpy(intt_table_, pool.intt_tables().data(), coeff_modulus * n * sizeof(Root), hipMemcpyHostToDevice);

    hipMalloc(&n_inverse_, coeff_modulus * sizeof(Ninverse));
    hipMemcpy(n_inverse_, pool.n_inverse().data(), coeff_modulus * sizeof(Ninverse), hipMemcpyHostToDevice);

    hipMalloc(&last_q_modinv_, (coeff_modulus - 1) * sizeof(Data));
    hipMemcpy(last_q_modinv_, pool.last_q_modinv().data(), (coeff_modulus - 1) * sizeof(Data), hipMemcpyHostToDevice);


    hipMalloc(&base_Bsk_, pool.base_Bsk().size() * sizeof(Modulus));
    hipMemcpy(base_Bsk_, pool.base_Bsk().data(), pool.base_Bsk().size() * sizeof(Modulus), hipMemcpyHostToDevice);

    hipMalloc(&bsk_ntt_tables_, pool.base_Bsk().size() * n * sizeof(Root));
    hipMemcpy(bsk_ntt_tables_, pool.bsk_ntt_tables().data(), pool.base_Bsk().size() * n * sizeof(Root), hipMemcpyHostToDevice);

    hipMalloc(&bsk_intt_tables_, pool.base_Bsk().size() * n * sizeof(Root));
    hipMemcpy(bsk_intt_tables_, pool.bsk_intt_tables().data(), pool.base_Bsk().size() * n * sizeof(Root), hipMemcpyHostToDevice);

    hipMalloc(&bsk_n_inverse_, pool.base_Bsk().size() * sizeof(Ninverse));
    hipMemcpy(bsk_n_inverse_, pool.bsk_n_inverse().data(), pool.base_Bsk().size() * sizeof(Ninverse), hipMemcpyHostToDevice);
    
    m_tilde_ = pool.m_tilde();

    hipMalloc(&base_change_matrix_Bsk_, pool.base_change_matrix_Bsk().size() * sizeof(Data));
    hipMemcpy(base_change_matrix_Bsk_, pool.base_change_matrix_Bsk().data(), pool.base_change_matrix_Bsk().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&inv_punctured_prod_mod_base_array_, pool.inv_punctured_prod_mod_base_array().size() * sizeof(Data));
    hipMemcpy(inv_punctured_prod_mod_base_array_, pool.inv_punctured_prod_mod_base_array().data(), pool.inv_punctured_prod_mod_base_array().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&base_change_matrix_m_tilde_, pool.base_change_matrix_m_tilde().size() * sizeof(Data));
    hipMemcpy(base_change_matrix_m_tilde_, pool.base_change_matrix_m_tilde().data(), pool.base_change_matrix_m_tilde().size() * sizeof(Data), hipMemcpyHostToDevice);

    inv_prod_q_mod_m_tilde_ = pool.inv_prod_q_mod_m_tilde();

    hipMalloc(&inv_m_tilde_mod_Bsk_, pool.inv_m_tilde_mod_Bsk().size() * sizeof(Data));
    hipMemcpy(inv_m_tilde_mod_Bsk_, pool.inv_m_tilde_mod_Bsk().data(), pool.inv_m_tilde_mod_Bsk().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&prod_q_mod_Bsk_, pool.prod_q_mod_Bsk().size() * sizeof(Data));
    hipMemcpy(prod_q_mod_Bsk_, pool.prod_q_mod_Bsk().data(), pool.prod_q_mod_Bsk().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&inv_prod_q_mod_Bsk_, pool.inv_prod_q_mod_Bsk().size() * sizeof(Data));
    hipMemcpy(inv_prod_q_mod_Bsk_, pool.inv_prod_q_mod_Bsk().data(), pool.inv_prod_q_mod_Bsk().size() * sizeof(Data), hipMemcpyHostToDevice);

    plain_modulus_ = pool.plain_modulus();

    hipMalloc(&base_change_matrix_q_, pool.base_change_matrix_q().size() * sizeof(Data));
    hipMemcpy(base_change_matrix_q_, pool.base_change_matrix_q().data(), pool.base_change_matrix_q().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&base_change_matrix_msk_, pool.base_change_matrix_msk().size() * sizeof(Data));
    hipMemcpy(base_change_matrix_msk_, pool.base_change_matrix_msk().data(), pool.base_change_matrix_msk().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&inv_punctured_prod_mod_B_array_, pool.inv_punctured_prod_mod_B_array().size() * sizeof(Data));
    hipMemcpy(inv_punctured_prod_mod_B_array_, pool.inv_punctured_prod_mod_B_array().data(), pool.inv_punctured_prod_mod_B_array().size() * sizeof(Data), hipMemcpyHostToDevice);

    inv_prod_B_mod_m_sk_ = pool.inv_prod_B_mod_m_sk();

    hipMalloc(&prod_B_mod_q_, pool.prod_B_mod_q().size() * sizeof(Data));
    hipMemcpy(prod_B_mod_q_, pool.prod_B_mod_q().data(), pool.prod_B_mod_q().size() * sizeof(Data), hipMemcpyHostToDevice);



    // For new
    hipMalloc(&q_Bsk_merge_modulus_, pool.q_Bsk_merge_modulus().size() * sizeof(Modulus));
    hipMemcpy(q_Bsk_merge_modulus_, pool.q_Bsk_merge_modulus().data(), pool.q_Bsk_merge_modulus().size() * sizeof(Modulus), hipMemcpyHostToDevice);
    
    hipMalloc(&q_Bsk_merge_ntt_tables_, pool.q_Bsk_merge_modulus().size() * n * sizeof(Root));
    hipMemcpy(q_Bsk_merge_ntt_tables_, pool.q_Bsk_merge_ntt_tables().data(), pool.q_Bsk_merge_modulus().size() * n * sizeof(Root), hipMemcpyHostToDevice);

    hipMalloc(&q_Bsk_merge_intt_tables_, pool.q_Bsk_merge_modulus().size() * n * sizeof(Root));
    hipMemcpy(q_Bsk_merge_intt_tables_, pool.q_Bsk_merge_intt_tables().data(), pool.q_Bsk_merge_modulus().size() * n * sizeof(Root), hipMemcpyHostToDevice);

    hipMalloc(&q_Bsk_n_inverse_, pool.q_Bsk_merge_modulus().size() * sizeof(Ninverse));
    hipMemcpy(q_Bsk_n_inverse_, pool.q_Bsk_n_inverse().data(), pool.q_Bsk_merge_modulus().size() * sizeof(Ninverse), hipMemcpyHostToDevice);


    half_ = pool.half();

    hipMalloc(&half_mod_, pool.half_mod().size() * sizeof(Data));
    hipMemcpy(half_mod_, pool.half_mod().data(), pool.half_mod().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&factor_, pool.factor().size() * sizeof(Data));
    hipMemcpy(factor_, pool.factor().data(), pool.factor().size() * sizeof(Data), hipMemcpyHostToDevice);


    hipMalloc(&plain_modulus2_, 1 * sizeof(Modulus));
    hipMemcpy(plain_modulus2_, pool.plain_modulus2().data(), 1 * sizeof(Modulus), hipMemcpyHostToDevice);

    hipMalloc(&n_plain_inverse_, 1 * sizeof(Ninverse));
    hipMemcpy(n_plain_inverse_, pool.n_plain_inverse().data(), 1 * sizeof(Ninverse), hipMemcpyHostToDevice);
    
    hipMalloc(&plain_ntt_tables_, n * sizeof(Root));
    hipMemcpy(plain_ntt_tables_, pool.plain_ntt_tables().data(), n * sizeof(Root), hipMemcpyHostToDevice);

    hipMalloc(&plain_intt_tables_, n * sizeof(Root));
    hipMemcpy(plain_intt_tables_, pool.plain_intt_tables().data(), n * sizeof(Root), hipMemcpyHostToDevice);



    gamma_ = pool.gamma();

    hipMalloc(&coeeff_div_plainmod_, pool.coeeff_div_plainmod().size() * sizeof(Data));
    hipMemcpy(coeeff_div_plainmod_, pool.coeeff_div_plainmod().data(), pool.coeeff_div_plainmod().size() * sizeof(Data), hipMemcpyHostToDevice);

    Q_mod_t_ = pool.Q_mod_t();

    upper_threshold_ = pool.upper_threshold();

    hipMalloc(&upper_halfincrement_, pool.upper_halfincrement().size() * sizeof(Data));
    hipMemcpy(upper_halfincrement_, pool.upper_halfincrement().data(), pool.upper_halfincrement().size() * sizeof(Data), hipMemcpyHostToDevice);



    hipMalloc(&Qi_t_, pool.Qi_t().size() * sizeof(Data));
    hipMemcpy(Qi_t_, pool.Qi_t().data(), pool.Qi_t().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&Qi_gamma_, pool.Qi_gamma().size() * sizeof(Data));
    hipMemcpy(Qi_gamma_, pool.Qi_gamma().data(), pool.Qi_gamma().size() * sizeof(Data), hipMemcpyHostToDevice);

    hipMalloc(&Qi_inverse_, pool.Qi_inverse().size() * sizeof(Data));
    hipMemcpy(Qi_inverse_, pool.Qi_inverse().data(), pool.Qi_inverse().size() * sizeof(Data), hipMemcpyHostToDevice);

    mulq_inv_t_ = pool.mulq_inv_t();
    mulq_inv_gamma_ = pool.mulq_inv_gamma();
    inv_gamma_ = pool.inv_gamma();


    /////////////////////////////////////////////////////////////////////////////
    
    // Encode -Decode Index
    std::vector<Data> encode_index;

    int m = n << 1;
    int gen = 3;
    int pos = 1;
    int index = 0;
    int location = 0;
    for (int i = 0; i < int(n / 2); i++) {

        index = (pos - 1) >> 1;
        location = bitreverse(index, n_power);
        //encoding_location_[i] = location;
        encode_index.push_back(location);
        pos *= gen;
        pos &= (m - 1);

    }
    for (int i = int(n / 2); i < n; i++) {

        index = (m - pos - 1) >> 1;
        location = bitreverse(index, n_power);
        //encoding_location_[i] = location;
        encode_index.push_back(location);
        pos *= gen;
        pos &= (m - 1);

    }

    hipMalloc(&encoding_location_, n * sizeof(Data));
    hipMemcpy(encoding_location_, encode_index.data(), n * sizeof(Data), hipMemcpyHostToDevice);

    /////////////////////////////////////////////////////////////////////////////


    hipMalloc(&temp1_enc, 3 * n * coeff_modulus * sizeof(Data));
    hipMalloc(&temp2_enc, 2 * n * coeff_modulus * sizeof(Data));


    hipMalloc(&temp1_mul, 4 * n * (bsk_modulus+(coeff_modulus - 1)) * sizeof(Data)); 
    hipMalloc(&temp2_mul, 3 * n * (bsk_modulus+(coeff_modulus - 1)) * sizeof(Data)); 
    hipMalloc(&temp3_mul, 3 * n * (coeff_modulus - 1) * sizeof(Data));


    hipMalloc(&temp1_relin, n * (coeff_modulus - 1) * coeff_modulus * sizeof(Data));
    hipMalloc(&temp2_relin, 2 * n * coeff_modulus * sizeof(Data));


    hipMalloc(&temp0_rotation, 2 * n * (coeff_modulus - 1) * sizeof(Data));
    hipMalloc(&temp1_rotation, n * (coeff_modulus - 1) * coeff_modulus * sizeof(Data));
    hipMalloc(&temp2_rotation, 2 * n * coeff_modulus * sizeof(Data));


    hipMalloc(&temp1_plain_mul, n * (coeff_modulus - 1) * sizeof(Data));


}

///////////////////////////////////////////////////////////////////////////



__host__ HEStream::HEStream(Parameters context){

    ring_size = context.n;
    coeff_modulus_count = context.coeff_modulus;
    bsk_modulus_count = context.bsk_modulus;

    hipStreamCreate(&stream); 

    hipMallocAsync(&temp1_enc, 3 * ring_size * coeff_modulus_count * sizeof(Data), stream);
    hipMallocAsync(&temp2_enc, 2 * ring_size * coeff_modulus_count * sizeof(Data), stream);

    hipMallocAsync(&temp1_mul, 4 * ring_size * (bsk_modulus_count+(coeff_modulus_count - 1)) * sizeof(Data), stream); 
    hipMallocAsync(&temp2_mul, 3 * ring_size * (bsk_modulus_count+(coeff_modulus_count - 1)) * sizeof(Data), stream); 
    //hipMallocAsync(&temp3_mul, 3 * ring_size * (coeff_modulus_count - 1) * sizeof(Data), stream);

    hipMallocAsync(&temp1_relin, ring_size * (coeff_modulus_count - 1) * coeff_modulus_count * sizeof(Data), stream);
    hipMallocAsync(&temp2_relin, 2 * ring_size * coeff_modulus_count * sizeof(Data), stream);

    hipMallocAsync(&temp0_rotation, 2 * ring_size * (coeff_modulus_count - 1) * sizeof(Data), stream);
    hipMallocAsync(&temp1_rotation, ring_size * (coeff_modulus_count - 1) * coeff_modulus_count * sizeof(Data), stream);
    hipMallocAsync(&temp2_rotation, 2 * ring_size * coeff_modulus_count * sizeof(Data), stream);

    hipMallocAsync(&temp1_plain_mul, ring_size * (coeff_modulus_count - 1) * sizeof(Data), stream);

}

__host__ void HEStream::kill(){

    hipStreamDestroy(stream);

    hipFree(temp1_enc);
    hipFree(temp2_enc);

    hipFree(temp1_mul);
    hipFree(temp2_mul);
    hipFree(temp3_mul);

    hipFree(temp1_relin);
    hipFree(temp2_relin);

    hipFree(temp0_rotation);
    hipFree(temp1_rotation);
    hipFree(temp2_rotation);

    hipFree(temp1_plain_mul);

}

///////////////////////////////////////////////////////////////////////////
 
__host__ Ciphertext::Ciphertext()
{

    ring_size = 0;
    coeff_modulus_count = 0;
    cipher_size = 0;

}

__host__ Ciphertext::Ciphertext(Parameters context)
{

    coeff_modulus_count = context.coeff_modulus - 1; 
    cipher_size = 3; //default
    ring_size = context.n; // n

    hipMalloc(&location, cipher_size * coeff_modulus_count * ring_size * sizeof(Data));

}

__host__ Ciphertext::Ciphertext(Parameters context, HEStream stream)
{

    coeff_modulus_count = context.coeff_modulus - 1; 
    cipher_size = 3; //default
    ring_size = context.n; // n

    hipMallocAsync(&location, cipher_size * coeff_modulus_count * ring_size * sizeof(Data), stream.stream);

}

__host__ Ciphertext::Ciphertext(Data* cipher, Parameters context)
{

    coeff_modulus_count = context.coeff_modulus - 1; 
    cipher_size = 3; //default
    ring_size = context.n; // n

    hipMalloc(&location, cipher_size * coeff_modulus_count * ring_size * sizeof(Data));
    hipMemcpy(location, cipher, cipher_size * coeff_modulus_count * ring_size * sizeof(Data), hipMemcpyHostToDevice);

}

__host__ Ciphertext::Ciphertext(Data* cipher, Parameters context, HEStream stream)
{

    coeff_modulus_count = context.coeff_modulus - 1; 
    cipher_size = 3; //default
    ring_size = context.n; // n

    hipMallocAsync(&location, cipher_size * coeff_modulus_count * ring_size * sizeof(Data), stream.stream);
    hipMemcpyAsync(location, cipher, cipher_size * coeff_modulus_count * ring_size * sizeof(Data), hipMemcpyHostToDevice, stream.stream);

}

__host__ void Ciphertext::kill()
{
    hipFree(location);
}

///////////////////////////////////////////////////////////////////////////

__host__ Message::Message()
{

    ring_size = 0;

}

__host__ Message::Message(Parameters context)
{

    ring_size = context.n; // n

    hipMalloc(&location, ring_size * sizeof(Data));

}


__host__ Message::Message(Parameters context, HEStream stream){

    ring_size = context.n; // n

    hipMallocAsync(&location, ring_size * sizeof(Data), stream.stream);

}

__host__ Message::Message(Data* message, Parameters context)
{

    ring_size = context.n; // n

    hipMalloc(&location, ring_size * sizeof(Data));
    hipMemcpy(location, message, ring_size * sizeof(Data), hipMemcpyHostToDevice);

}

__host__ Message::Message(const std::vector<uint64_t> &message, Parameters context)
{

    ring_size = context.n; // n

    hipMalloc(&location, ring_size * sizeof(Data));
    hipMemcpy(location, message.data(), ring_size * sizeof(Data), hipMemcpyHostToDevice);

}

 __host__ Message::Message(Data* message, Parameters context, HEStream stream){

    ring_size = context.n; // n

    hipMallocAsync(&location, ring_size * sizeof(Data), stream.stream);
    hipMemcpyAsync(location, message, ring_size * sizeof(Data), hipMemcpyHostToDevice, stream.stream);

 }

__host__ Message::Message(Data* message, int size, Parameters context)
{

    ring_size = context.n; // n

    hipMalloc(&location, ring_size * sizeof(Data));
    hipMemcpy(location, message, size * sizeof(Data), hipMemcpyHostToDevice);

}

__host__ Message::Message(const std::vector<uint64_t> &message, int size, Parameters context)
{

    ring_size = context.n; // n

    hipMalloc(&location, ring_size * sizeof(Data));
    hipMemcpy(location, message.data(), size * sizeof(Data), hipMemcpyHostToDevice);

}

__host__ Message::Message(Data* message, int size, Parameters context, HEStream stream){

    ring_size = context.n; // n

    hipMallocAsync(&location, ring_size * sizeof(Data), stream.stream);
    hipMemcpyAsync(location, message, size * sizeof(Data), hipMemcpyHostToDevice, stream.stream);

}

__host__ void Message::kill()
{
    hipFree(location);
}

///////////////////////////////////////////////////////////////////////////

__host__ Plaintext::Plaintext()
{

    ring_size = 0;

}

__host__ Plaintext::Plaintext(Parameters context)
{

    ring_size = context.n; // n

    hipMalloc(&location, ring_size * sizeof(Data));

}

__host__ Plaintext::Plaintext(Parameters context, HEStream stream){

    ring_size = context.n; // n

    hipMallocAsync(&location, ring_size * sizeof(Data), stream.stream);

}

__host__ Plaintext::Plaintext(Message message, Parameters context)
{

    ring_size = context.n; // n

    hipMalloc(&location, ring_size * sizeof(Data));
    hipMemcpy(location, message.location, ring_size * sizeof(Data), hipMemcpyHostToDevice);

}

__host__ Plaintext::Plaintext(Message message, Parameters context, HEStream stream){

    ring_size = context.n; // n

    hipMallocAsync(&location, ring_size * sizeof(Data), stream.stream);
    hipMemcpyAsync(location, message.location, ring_size * sizeof(Data), hipMemcpyHostToDevice, stream.stream);

}

__host__ void Plaintext::kill()
{
    hipFree(location);
}




///////////////////////////////////////////////////////////////////////////

__host__ Relinkey::Relinkey()
{

    ring_size = 0;
    coeff_modulus_count = 0;

}

__host__ Relinkey::Relinkey(Parameters context)
{

    coeff_modulus_count = context.coeff_modulus - 1; 
    ring_size = context.n; // n

    hipMalloc(&location, 2 * coeff_modulus_count * (coeff_modulus_count + 1) * ring_size * sizeof(Data));

    hipMalloc(&e_a, 2 * (coeff_modulus_count + 1) * ring_size * sizeof(Data));
    
}

__host__ void Relinkey::kill()
{
    hipFree(location);
    hipFree(e_a);
}

///////////////////////////////////////////////////////////////////////////

__host__ Galoiskey::Galoiskey()
{

    ring_size = 0;
    coeff_modulus_count = 0;

}

__host__ Galoiskey::Galoiskey(Parameters context)
{

    coeff_modulus_count = context.coeff_modulus - 1; 
    ring_size = context.n; // n

    galois_elt_pos = (int*)malloc(MAX_SHIFT * sizeof(int));
    galois_elt_neg = (int*)malloc(MAX_SHIFT * sizeof(int));

    for(int i = 0; i < MAX_SHIFT; i++){
        hipMalloc(&positive_location[i], 2 * coeff_modulus_count * (coeff_modulus_count + 1) * ring_size * sizeof(Data));
        hipMalloc(&negative_location[i], 2 * coeff_modulus_count * (coeff_modulus_count + 1) * ring_size * sizeof(Data));
    }
    
    hipMalloc(&e_a, 2 * (coeff_modulus_count + 1) * ring_size * sizeof(Data));
    
}

__host__ void Galoiskey::kill()
{   
    for(int i = 0; i < MAX_SHIFT; i++){
        hipFree(positive_location[i]);
        hipFree(negative_location[i]);
    }

    free(galois_elt_pos);
    free(galois_elt_neg);

    hipFree(e_a);
}


///////////////////////////////////////////////////////////////////////////

__host__ Secretkey::Secretkey()
{

    ring_size = 0;
    coeff_modulus_count = 0;

}

__host__ Secretkey::Secretkey(Parameters context)
{

    coeff_modulus_count = context.coeff_modulus;
    ring_size = context.n; // n

    hipMalloc(&location, coeff_modulus_count * ring_size * sizeof(Data));

}

__host__ void Secretkey::kill()
{
    hipFree(location);
}

///////////////////////////////////////////////////////////////////////////

__host__ Publickey::Publickey()
{

    ring_size = 0;
    coeff_modulus_count = 0;

}

__host__ Publickey::Publickey(Parameters context)
{

    coeff_modulus_count = context.coeff_modulus;
    ring_size = context.n; // n

    hipMalloc(&location, 2 * coeff_modulus_count * ring_size * sizeof(Data));

}

__host__ void Publickey::kill()
{
    hipFree(location);
}